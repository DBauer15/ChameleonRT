#include "hip/hip_runtime.h"
#include <hip/hip_math_constants.h>
#include <optix.h>
#include <optix_math.h>

typedef unsigned long long uint64_t;
typedef unsigned int uint32_t;

// Global camera parameters
rtDeclareVariable(float3, cam_pos, , );
rtDeclareVariable(float3, cam_du, , );
rtDeclareVariable(float3, cam_dv, , );
rtDeclareVariable(float3, cam_dir_top_left, , );

rtDeclareVariable(rtObject, scene, , );

rtDeclareVariable(uint2, pixel, rtLaunchIndex, );
rtDeclareVariable(uint2, screen, rtLaunchDim, );

rtBuffer<uchar4, 2> framebuffer;
rtBuffer<float4, 2> accum_buffer;

// View params buffer:
// camera position, dir_du, dir_dv, dir_top_left
struct ViewParams {
	float4 cam_pos;
	float4 cam_du;
	float4 cam_dv;
	float4 cam_dir_top_left;
	uint32_t frame_id;
};
rtBuffer<ViewParams, 1> view_params;

rtBuffer<int3, 1> index_buffer;
rtBuffer<float3, 1> vertex_buffer;

struct RayPayload {
	// float3 color, float depth
	float4 color_dist;
	// float3 normal, float 1/0 if occlusion hit
	float4 normal_hit;
};

__device__ RayPayload make_ray_payload() {
	RayPayload p;
	p.color_dist = make_float4(0.f);
	p.normal_hit = make_float4(0.f);
	return p;
}

rtDeclareVariable(RayPayload, ray_payload, rtPayload, );

// http://www.pcg-random.org/download.html
struct PCGRand {
	uint64_t state;
	// Just use stream 1
};

__device__ uint32_t pcg32_random(PCGRand &rng) {
	uint64_t oldstate = rng.state;
	rng.state = oldstate * 6364136223846793005ULL + 1;
	// Calculate output function (XSH RR), uses old state for max ILP
	uint32_t xorshifted = ((oldstate >> 18u) ^ oldstate) >> 27u;
	uint32_t rot = oldstate >> 59u;
	return (xorshifted >> rot) | (xorshifted << ((-rot) & 31));
}

__device__ float pcg32_randomf(PCGRand &rng) {
	return ldexp((double)pcg32_random(rng), -32);
}

__device__ PCGRand get_rng(uint32_t frame_id) {
	uint32_t seed = (pixel.x + pixel.y * screen.x) * (frame_id + 1);

	PCGRand rng;
	rng.state = 0;
	pcg32_random(rng);
	rng.state += seed;
	pcg32_random(rng);
	return rng;
}

__device__ float linear_to_srgb(float x) {
	if (x <= 0.0031308f) {
		return 12.92f * x;
	}
	return 1.055f * pow(x, 1.f/2.4f) - 0.055f;
}

__device__ void ortho_basis(float3 &v_x, float3 &v_y, const float3 &n) {
	v_y = make_float3(0.f, 0.f, 0.f);

	if (n.x < 0.6f && n.x > -0.6f) {
		v_y.x = 1.f;
	} else if (n.y < 0.6f && n.y > -0.6f) {
		v_y.y = 1.f;
	} else if (n.z < 0.6f && n.z > -0.6f) {
		v_y.z = 1.f;
	} else {
		v_y.x = 1.f;
	}
	v_x = normalize(cross(v_y, n));
	v_y = normalize(cross(n, v_x));
}

RT_PROGRAM void perspective_camera() {
	const int PRIMARY_RAY = 0;
	const int OCCLUSION_RAY = 1;

	const ViewParams view = view_params[0];
	PCGRand rng = get_rng(view.frame_id);
	const float2 d = make_float2(pixel.x + pcg32_randomf(rng), pixel.y + pcg32_randomf(rng)) / make_float2(screen);
	const float3 ray_dir = normalize(d.x * make_float3(view.cam_du)
			+ d.y * make_float3(view.cam_dv) + make_float3(view.cam_dir_top_left));

	optix::Ray ray(make_float3(view.cam_pos), ray_dir, PRIMARY_RAY, 0.0001);

	RayPayload payload = make_ray_payload();
	rtTrace(scene, ray, payload, RT_VISIBILITY_ALL,
			RTrayflags(RT_RAY_FLAG_DISABLE_ANYHIT));

	if (payload.color_dist.w > 0) {
		const float3 hit_p = ray.origin + payload.color_dist.w * ray.direction;
		float3 v_x, v_y;
		float3 v_z = make_float3(payload.normal_hit);
		ortho_basis(v_x, v_y, v_z);

		// Sample the hemisphere
		const float theta = sqrt(pcg32_randomf(rng));
		const float phi = 2.0f * HIP_PI_F * pcg32_randomf(rng);

		const float x = cos(phi) * theta;
		const float y = sin(phi) * theta;
		const float z = sqrt(1.0 - theta * theta);

		float3 ao_dir;
		ao_dir.x = x * v_x.x + y * v_y.x + z * v_z.x;
		ao_dir.y = x * v_x.y + y * v_y.y + z * v_z.y;
		ao_dir.z = x * v_x.z + y * v_y.z + z * v_z.z;

		optix::Ray ao_ray(hit_p, ao_dir, OCCLUSION_RAY, 0.0001);
		RayPayload ao_payload = make_ray_payload();
		rtTrace(scene, ao_ray, ao_payload, RT_VISIBILITY_ALL,
				RTrayflags(RT_RAY_FLAG_TERMINATE_ON_FIRST_HIT));
		if (ao_payload.normal_hit.w != 0.f) {
			payload.color_dist = make_float4(0.f);
		}
	}
	// TODO: Math operations for the built in floatN types

	const float4 accum_color = (payload.color_dist + view.frame_id * accum_buffer[pixel]) / (view.frame_id + 1);
	accum_buffer[pixel] = accum_color;

	framebuffer[pixel] = make_uchar4(linear_to_srgb(accum_color.x) * 255.f,
			linear_to_srgb(accum_color.y) * 255.f,
			linear_to_srgb(accum_color.z) * 255.f, 255);
}

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

RT_PROGRAM void closest_hit() {
	const int3 indices = index_buffer[rtGetPrimitiveIndex()];
	const float3 v0 = vertex_buffer[indices.x];
	const float3 v1 = vertex_buffer[indices.y];
	const float3 v2 = vertex_buffer[indices.z];
	const float3 normal = normalize(cross(v1 - v0, v2 - v0));
	const float3 color = (normal + make_float3(1.f)) * 0.5f;
	ray_payload.color_dist = make_float4(color.x, color.y, color.z, t_hit);
	ray_payload.normal_hit = make_float4(normal, 1.f);
}

RT_PROGRAM void occlusion_hit() {
	ray_payload.normal_hit.w = 1;
}

