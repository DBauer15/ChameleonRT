#include "hip/hip_runtime.h"
#include <hip/hip_math_constants.h>
#include <optix.h>
#include <optix_math.h>

#ifndef M_1_PI
#define M_1_PI 0.318309886183790671538
#endif

typedef unsigned long long uint64_t;
typedef unsigned int uint32_t;

// Global camera parameters
rtDeclareVariable(float3, cam_pos, , );
rtDeclareVariable(float3, cam_du, , );
rtDeclareVariable(float3, cam_dv, , );
rtDeclareVariable(float3, cam_dir_top_left, , );

rtDeclareVariable(rtObject, scene, , );

rtDeclareVariable(uint2, pixel, rtLaunchIndex, );
rtDeclareVariable(uint2, screen, rtLaunchDim, );

rtBuffer<uchar4, 2> framebuffer;
rtBuffer<float4, 2> accum_buffer;

// View params buffer:
// camera position, dir_du, dir_dv, dir_top_left
struct ViewParams {
	float4 cam_pos;
	float4 cam_du;
	float4 cam_dv;
	float4 cam_dir_top_left;
	uint32_t frame_id;
};
rtBuffer<ViewParams, 1> view_params;

rtBuffer<int3, 1> index_buffer;
rtBuffer<float3, 1> vertex_buffer;

struct RayPayload {
	// float3 color, float depth
	float4 color_dist;
	// float3 normal, float 1/0 if occlusion hit
	float4 normal_hit;
};

__device__ RayPayload make_ray_payload() {
	RayPayload p;
	p.color_dist = make_float4(0.f);
	p.normal_hit = make_float4(0.f);
	return p;
}

rtDeclareVariable(RayPayload, ray_payload, rtPayload, );

// http://www.pcg-random.org/download.html
struct PCGRand {
	uint64_t state;
	// Just use stream 1
};

__device__ uint32_t pcg32_random(PCGRand &rng) {
	uint64_t oldstate = rng.state;
	rng.state = oldstate * 6364136223846793005ULL + 1;
	// Calculate output function (XSH RR), uses old state for max ILP
	uint32_t xorshifted = ((oldstate >> 18u) ^ oldstate) >> 27u;
	uint32_t rot = oldstate >> 59u;
	return (xorshifted >> rot) | (xorshifted << ((-rot) & 31));
}

__device__ float pcg32_randomf(PCGRand &rng) {
	return ldexp((double)pcg32_random(rng), -32);
}

__device__ PCGRand get_rng(uint32_t frame_id) {
	uint32_t seed = (pixel.x + pixel.y * screen.x) * (frame_id + 1);

	PCGRand rng;
	rng.state = 0;
	pcg32_random(rng);
	rng.state += seed;
	pcg32_random(rng);
	return rng;
}

__device__ float linear_to_srgb(float x) {
	if (x <= 0.0031308f) {
		return 12.92f * x;
	}
	return 1.055f * pow(x, 1.f/2.4f) - 0.055f;
}

__device__ void ortho_basis(float3 &v_x, float3 &v_y, const float3 &n) {
	v_y = make_float3(0.f, 0.f, 0.f);

	if (n.x < 0.6f && n.x > -0.6f) {
		v_y.x = 1.f;
	} else if (n.y < 0.6f && n.y > -0.6f) {
		v_y.y = 1.f;
	} else if (n.z < 0.6f && n.z > -0.6f) {
		v_y.z = 1.f;
	} else {
		v_y.x = 1.f;
	}
	v_x = normalize(cross(v_y, n));
	v_y = normalize(cross(n, v_x));
}

RT_PROGRAM void perspective_camera() {
	const int PRIMARY_RAY = 0;
	const int OCCLUSION_RAY = 1;
	const int MAX_PATH_DEPTH = 5;

	const ViewParams view = view_params[0];
	PCGRand rng = get_rng(view.frame_id);
	const float2 d = make_float2(pixel.x + pcg32_randomf(rng), pixel.y + pcg32_randomf(rng)) / make_float2(screen);
	const float3 ray_dir = normalize(d.x * make_float3(view.cam_du)
			+ d.y * make_float3(view.cam_dv) + make_float3(view.cam_dir_top_left));

	optix::Ray ray(make_float3(view.cam_pos), ray_dir, PRIMARY_RAY, 0.0);

	const float3 light_emission = make_float3(1.0);
	int bounce = 0;
	float3 illum = make_float3(0.0);
	float3 path_throughput = make_float3(1.0);
	do {
		RayPayload payload = make_ray_payload();
		rtTrace(scene, ray, payload, RT_VISIBILITY_ALL,
				RTrayflags(RT_RAY_FLAG_DISABLE_ANYHIT));

		if (payload.color_dist.w <= 0) {
			break;
		}

		const float3 hit_p = ray.origin + payload.color_dist.w * ray.direction;
		float3 v_x, v_y;
		float3 v_z = make_float3(payload.normal_hit);
		ortho_basis(v_x, v_y, v_z);

		const float3 bsdf = make_float3(payload.color_dist) * M_1_PI;

		// Direct light sampling.
		const float3 w_o = -ray.direction;
		const float3 light_dir = normalize(make_float3(-0.5, 0.8, 0.5));

		optix::Ray shadow_ray(hit_p, light_dir, OCCLUSION_RAY, 0.0001);
		RayPayload shadow_payload = make_ray_payload();
		rtTrace(scene, shadow_ray, shadow_payload, RT_VISIBILITY_ALL,
				RTrayflags(RT_RAY_FLAG_TERMINATE_ON_FIRST_HIT | RT_RAY_FLAG_DISABLE_ANYHIT));
		if (shadow_payload.normal_hit.w == 0.f) {
			illum += path_throughput * bsdf * light_emission * abs(dot(light_dir, v_z));
		}

		// Sample the hemisphere
		// TODO: Cosine weighted hemisphere sampling
		const float theta = sqrt(pcg32_randomf(rng));
		const float phi = 2.0f * HIP_PI_F * pcg32_randomf(rng);

		const float x = cos(phi) * theta;
		const float y = sin(phi) * theta;
		const float z = sqrt(1.0 - theta * theta);

		float3 w_i;
		w_i.x = x * v_x.x + y * v_y.x + z * v_z.x;
		w_i.y = x * v_x.y + y * v_y.y + z * v_z.y;
		w_i.z = x * v_x.z + y * v_y.z + z * v_z.z;
		w_i = normalize(w_i);

		// Update path throughput and continue the ray
		// TODO: This is just a hard-coded Lambertian BRDF,
		// using the object's normal color as its albedo
		float pdf = abs(dot(w_i, v_z)) * M_1_PI;
		if (pdf == 0.0) {
			break;
		}
		// Note: same as just multiplying my M_PI b/c the cancellation,
		// but left like this b/c I'll swap to Disney BRDF soon-ish
		path_throughput *= bsdf * abs(dot(w_i, v_z)) / pdf;
		if (path_throughput.x == 0 && path_throughput.y == 0 && path_throughput.z == 0) {
			break;
		}

		// Update ray
		ray.origin = hit_p;
		ray.direction = w_i;
		ray.tmin = 0.0001;
		ray.tmax = 1e20f;

		++bounce;
	} while (bounce < MAX_PATH_DEPTH);

	const float4 accum_color = (make_float4(illum, 1.0) + view.frame_id * accum_buffer[pixel]) / (view.frame_id + 1);
	accum_buffer[pixel] = accum_color;

	framebuffer[pixel] = make_uchar4(linear_to_srgb(accum_color.x) * 255.f,
			linear_to_srgb(accum_color.y) * 255.f,
			linear_to_srgb(accum_color.z) * 255.f, 255);
}

rtDeclareVariable(float, t_hit, rtIntersectionDistance, );

RT_PROGRAM void closest_hit() {
	const int3 indices = index_buffer[rtGetPrimitiveIndex()];
	const float3 v0 = vertex_buffer[indices.x];
	const float3 v1 = vertex_buffer[indices.y];
	const float3 v2 = vertex_buffer[indices.z];
	const float3 normal = normalize(cross(v1 - v0, v2 - v0));
	ray_payload.color_dist = make_float4(0.9, 0.9, 0.9, t_hit);
	ray_payload.normal_hit = make_float4(normal, 1.f);
}

RT_PROGRAM void occlusion_hit() {
	ray_payload.normal_hit.w = 1;
}

